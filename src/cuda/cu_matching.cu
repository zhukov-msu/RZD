
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ static inline unsigned int argmin(unsigned int a, unsigned int b, unsigned int c)
{ 
    if (a < b)
    {
        if (a < c) 
             return 0; 
        else 
             return 2;
    }

    if (b < c)
        return 1;
    else return 2;
}

__global__ void matching(int *D, int *phi, int m, int n)
{
    // matrix m x n
    int pnt = 2;
    const int tid = threadIdx.x;// + blockDim.x * blockIdx.x;
    if (tid == 0)
    {
        printf("%d", tid);
        int j = 0;
        for (j = 0; j < n; j++)
        {
            int tmp[3] = { D[(tid * n) + j], D[(tid+1)*n+j] + pnt, D[(tid * n) + j + 1] + pnt };
            int arg = argmin(tmp[0], tmp[1], tmp[2]);
            int dmin = tmp[arg];
            D[((tid+1) * n) + j + 1] = D[((tid+1) * n) + j + 1] + dmin;
            phi[(tid * n) + j] = arg + 1;
        }
    }
    else
    if (tid < m)
    {
        int j = 0;
        for (j = 0; j < n; j++)
        {
            while(1)
            {
                if (phi[(tid-1) * n + j])
                {
                    int tmp[3] = {D[(tid * n) + j], D[(tid+1)*n+j] + pnt, D[(tid * n) + j + 1] + pnt};
                    int arg = argmin(tmp[0], tmp[1], tmp[2]);
                    int dmin = tmp[arg];
                    D[((tid+1) * n) + j + 1] = D[((tid+1) * n) + j + 1] + dmin;
                    phi[(tid * n) + j] = arg + 1;
                    break;
                }
            }
        }
    }
}