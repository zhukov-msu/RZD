
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>
 
__device__ static inline unsigned int argmin(unsigned int a, unsigned int b, unsigned int c)
{ 
    if (a < b)
    {
        if (a < c) 
             return 0; 
        else 
             return 2;
    }

    if (b < c)
        return 1;
    else return 2;
}

__global__ void matching(int *D, unsigned int *phi, int m, int n)
{
    // matrix m x n
    int pnt = 2;
    
    const int tid = threadIdx.x;// + blockDim.x * blockIdx.x;

    if (tid < m)
    {
        int j = 0;
        int reserve = -threadIdx.x + 1;
        for (j = reserve; j < n; j++)
        {   
            if (j >= 1){
                int tmp[3] = {D[(tid * (n+1)) + j], D[(tid * (n+1)) + j + 1]+pnt, D[(tid+1)*(n+1)+j]+pnt};
                int arg = argmin(tmp[0], tmp[1], tmp[2]);
                int dmin = tmp[arg];
                D[(tid+1)*(n+1)+j+1] = D[(tid+1)*(n+1)+j+1] + dmin;
                phi[tid * n + j] = arg + 1;
            }
            __syncthreads();
        }
    }
}

// if (tid == 2){
//     printf("j: %d\n", j);
//     printf("idx: %d %d %d %d\n",(tid * (n+1)) + j + 1, (tid * (n+1)) + j + 2, (tid+1)*(n+1)+j+1, (tid+1)*(n+1)+j+2);
//     printf("tmp: %d %d %d\n",D[(tid * (n+1)) + j], D[(tid * (n+1)) + j + 1], D[(tid+1)*(n+1)+j]);
//     // printf("min: %d\n", tmp[arg]);
//     printf("D[i+1][j+1]: %d\n", D[(tid+1)*(n+1)+j+1]);
// }